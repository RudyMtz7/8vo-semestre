// Dot Priduct
// By
// Victor Hugo Torres Rivera
// A01701017
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define NTHREADS 200
#define NBLOCKS 200


__global__ void mat_mult(int *a_mat, int *b_mat, int *r_mat, int a_cols, int a_rows, int b_cols, int b_rows){
  // Get x and y
  int tidx = threadIdx.x + blockIdx.x* blockDim.x;
  int tidy = threadIdx.y + blockIdx.y* blockDim.y;
  // Calculate linear index
  int index = tidx + tidy * a_rows;
  int actual_row = 0;
  int actual_col = 0;
  int aux = 0;
  // If it fits the result matrix calculate
  if(index < b_cols * a_rows){
    // Calculate the actual indexes for a col and b row
    actual_col = index%b_cols;
    actual_row = index/b_cols;
    // Calculate the actual
    for (int i = 0; i < a_cols; i++){
      aux += a_mat[actual_col * a_cols + i] * b_mat [actual_row + b_cols * i];
    }
    r_mat[index] = aux;
  }
}



int main(void){
  // Device Matirxes
  int *d_a_mat, *d_b_mat, *d_r_mat;
  // Host Matrixes
  int *a_mat, *b_mat, *r_mat;
  // Matrixes Size
  int a_cols, b_cols, a_rows, b_rows;


  // Define Matrixes Sizes
  a_cols = 3;
  a_rows = 2;

  b_cols = 2;
  b_rows = a_cols;

  // Matrix Allocations
  // HOST
  a_mat = (int *)malloc(sizeof(int) * a_cols * a_rows);
  b_mat = (int *)malloc(sizeof(int) * b_cols * b_rows);
  r_mat = (int *)malloc(sizeof(int) * b_cols * a_rows);

  // Device
  hipMalloc((void **)&d_a_mat, sizeof(int) * a_cols * a_rows);
  hipMalloc((void **)&d_b_mat, sizeof(int) * b_cols * b_rows);
  hipMalloc((void **)&d_r_mat, sizeof(int) * b_cols * a_rows);

  srand(time(NULL));
  // Fill and Print Matrix A
  printf("A = ");
  for (int i = 0; i < a_cols * a_rows; i ++ ){
    if (i%a_cols == 0){
      printf("\n");
    }
    a_mat[i] = rand() % 20 - 10;
    printf("%i\t", a_mat[i]);
  }

  // Fill And Print Matrix B
  printf("\nB = ");
  for (int i = 0; i < b_cols * b_rows; i++){
    if (i%b_cols == 0){
      printf("\n");
    }
    b_mat[i] = rand() % 20 - 10;
    printf("%i\t", b_mat[i]);
  }


  // Pass values to device
  hipMemcpy(d_a_mat, a_mat, sizeof(int) * a_cols * a_rows, hipMemcpyHostToDevice);
  hipMemcpy(d_b_mat, b_mat, sizeof(int) * b_cols * b_rows, hipMemcpyHostToDevice);

  dim3 dimThreads(NTHREADS, 1, 1);
  dim3 dimBlocks(NBLOCKS, 1, 1);

  // Call Kernel function
  mat_mult<<<dimBlocks, dimThreads>>>(d_a_mat, d_b_mat, d_r_mat, a_cols, a_rows, b_cols, b_rows);

  hipMemcpy(r_mat, d_r_mat, sizeof(int) * a_rows * b_cols, hipMemcpyDeviceToHost);

  // Print Result Matrix
  printf("\nR = ");
  for (int i = 0; i < b_cols * a_rows; i++){
    if (i%b_cols == 0){
      printf("\n");
    }
    printf("%i\t", r_mat[i]);
  }
  printf("\n");

  // Free Memory
  free(a_mat);
  free(b_mat);
  free(r_mat);
  hipFree(d_a_mat);
  hipFree(d_b_mat);
  hipFree(d_r_mat);
}
