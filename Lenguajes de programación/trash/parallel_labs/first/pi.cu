// Pi Calculation
// By Victor Hugo Torres
// A01701017
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void pi_calc(double *sum, long nrects, double width, int  nblocks){
  int tid = blockIdx.x;
	double mid;
  int i;
	for (i=tid; i< nrects; i+=nblocks) {
		mid = (i+0.5)*width;
		sum[tid] += 4.0/(1.0+mid*mid);
	}
}


int main(void) {
  long rects  = 1000000000;
  int blocks = 200;
	double *sum, *d_sum;
	double width = 1.0/rects;
  double pi = 0;
  int index, auxindex;

  sum = (double *)malloc(blocks*sizeof(double));
  hipMalloc((void **) &d_sum, blocks*sizeof(double));


  for (auxindex = 0; auxindex < blocks; auxindex++){
    sum[auxindex] = 0;
  }

	hipMemcpy(d_sum, sum, blocks*sizeof(double), hipMemcpyHostToDevice);

	pi_calc <<<blocks,1>>> (d_sum, rects, width, blocks);

	hipMemcpy(sum, d_sum, blocks*sizeof(double), hipMemcpyDeviceToHost);

	for(index=0; index<blocks; index++){
		pi += sum[index];
  }

	pi *= width;


	printf("PI = %f\n",pi);


	free(sum);
	hipFree(d_sum);

	return 0;
}
