//Rodolfo Martínez Guevara A01700309

#include <stdio.h>
#include <hip/hip_runtime.h>


//Define constant variables for threads per block, total rectangles, total
//blocks
#define THREADS_PER_BLOCK 512
#define RECTANGLES 1000000
#define BLOCKS 1000


//Cuda function to calculate sum in device
__global__ void calculatePi(double *sum, double width){
  //Get current total index value
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	double mid, height;
  //Interal iteration value
  int i = tid;
  while(i < RECTANGLES){
    mid = (i + 0.5) * width;
    height = 4.0 / (1.0 + mid * mid);
    sum[tid] += height;
    //Skips total thread ammount so each block carries out the required
    //operations
    i = i + BLOCKS*THREADS_PER_BLOCK;
  }
}


int main(void) {
  //Define variables
	double *sum;
	double *sumAux;
  double width, area;
  width = 1.0/ (double) RECTANGLES;
  double aux = 0;

  //Create arrays for sum operations
  sum = (double *)malloc(BLOCKS*THREADS_PER_BLOCK*sizeof(double));
  //Device array for operation
  hipMalloc((void **) &sumAux, BLOCKS*THREADS_PER_BLOCK*sizeof(double));

  //Copy array to device array variable with given length of total threads
	hipMemcpy(sumAux, sum, BLOCKS*THREADS_PER_BLOCK*sizeof(double), hipMemcpyHostToDevice);

  //Call cuda function with given blocks and threads
	calculatePi<<<BLOCKS, THREADS_PER_BLOCK>>> (sumAux, width);

  //Copy values obtained from cuda function
	hipMemcpy(sum, sumAux, BLOCKS*THREADS_PER_BLOCK*sizeof(double), hipMemcpyDeviceToHost);

  //Returns total sum in a single variable
  for(int i=0; i < BLOCKS*THREADS_PER_BLOCK; i++){
		aux += sum[i];
  }
  // printf("sum = %lf\n",aux);
  //Calculates Pi
	area = width * aux;

	printf("Pi: %lf\n",area);

  free(sum);
	hipFree(sumAux);

	return 0;
}
