
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define NTHREADS 200
#define NBLOCKS 200


__global__ void multiplication(int *a_mat, int *b_mat, int *r_mat, int a_cols, int a_rows, int b_cols, int b_rows){
  int tidx = threadIdx.x + blockIdx.x* blockDim.x;
  int tidy = threadIdx.y + blockIdx.y* blockDim.y;
  int index = tidx + tidy * a_rows;
  int actual_row = 0;
  int actual_col = 0;
  int aux = 0;
  if(index < b_cols * a_rows){
    actual_col = index%b_cols;
    actual_row = index/b_cols;
    for (int i = 0; i < a_cols; i++){
      aux += a_mat[actual_col * a_cols + i] * b_mat [actual_row + b_cols * i];
    }
    r_mat[index] = aux;
  }
}



int main(void){
  int *d_a_mat, *d_b_mat, *d_r_mat;
  int *a_mat, *b_mat, *r_mat;
  int a_cols, b_cols, a_rows, b_rows;


  a_cols = 3;
  a_rows = 2;

  b_cols = 2;
  b_rows = a_cols;

  a_mat = (int *)malloc(sizeof(int) * a_cols * a_rows);
  b_mat = (int *)malloc(sizeof(int) * b_cols * b_rows);
  r_mat = (int *)malloc(sizeof(int) * b_cols * a_rows);

  hipMalloc((void **)&d_a_mat, sizeof(int) * a_cols * a_rows);
  hipMalloc((void **)&d_b_mat, sizeof(int) * b_cols * b_rows);
  hipMalloc((void **)&d_r_mat, sizeof(int) * b_cols * a_rows);

  srand(time(NULL));
  printf("A = ");
  for (int i = 0; i < a_cols * a_rows; i ++ ){
    if (i%a_cols == 0){
      printf("\n");
    }
    a_mat[i] = rand() % 20 - 10;
    printf("%i\t", a_mat[i]);
  }

  printf("\nB = ");
  for (int i = 0; i < b_cols * b_rows; i++){
    if (i%b_cols == 0){
      printf("\n");
    }
    b_mat[i] = rand() % 20 - 10;
    printf("%i\t", b_mat[i]);
  }


  hipMemcpy(d_a_mat, a_mat, sizeof(int) * a_cols * a_rows, hipMemcpyHostToDevice);
  hipMemcpy(d_b_mat, b_mat, sizeof(int) * b_cols * b_rows, hipMemcpyHostToDevice);

  dim3 dimThreads(NTHREADS, 1, 1);
  dim3 dimBlocks(NBLOCKS, 1, 1);

  multiplication<<<dimBlocks, dimThreads>>>(d_a_mat, d_b_mat, d_r_mat, a_cols, a_rows, b_cols, b_rows);

  hipMemcpy(r_mat, d_r_mat, sizeof(int) * a_rows * b_cols, hipMemcpyDeviceToHost);

  printf("\nR = ");
  for (int i = 0; i < b_cols * a_rows; i++){
    if (i%b_cols == 0){
      printf("\n");
    }
    printf("%i\t", r_mat[i]);
  }
  printf("\n");

  free(a_mat);
  free(b_mat);
  free(r_mat);
  hipFree(d_a_mat);
  hipFree(d_b_mat);
  hipFree(d_r_mat);
}
