//Señor Doctor Profesor Rodolfo Martínez Guevara
//A01700309

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define N_THREADS 200
#define N_BLOCKS 200


__global__ void multiplication(int *matrix_a, int *matrix_b, int *result_matrix, int cols_a, int rows_a, int cols_b, int rows_b){
  //Generate the index for each dimension of the matrix
  int tidx = threadIdx.x + blockIdx.x* blockDim.x;
  //Clean :^)
  int tidy = threadIdx.y + blockIdx.y* blockDim.y;
  //Get the global index
  int index = tidx + tidy * rows_a;
  int current_row = 0;
  int current_col = 0;
  int aux = 0;
  //While the index fits the resultant matrix
  if(index < cols_b * rows_a){
    //Generate the index of the result matrix
    current_col = index/cols_b;
    current_row = index%cols_b;
    //Calculate value for this position
    for (int i = 0; i < cols_a; i++){
      aux += matrix_a[current_col * cols_a + i] * matrix_b [current_row + cols_b * i];
    }
    result_matrix[index] = aux;
  }
}



int main(void){
  //Matrixes in device
  int *d_matrix_a, *d_matrix_b, *d_result_matrix;
  //Matrixes in CPU
  int *matrix_a, *matrix_b, *result_matrix;
  //Number of cols and rows for each matrix
  int cols_a, cols_b, rows_a, rows_b;
  //Auxiliar variables
  int aux_a, aux_b;

  //Get matrix dimensions
  printf("Cols in A: \n");
  scanf("%i", &cols_a);
  printf("Rows in A: \n");
  scanf("%i", &rows_a);

  printf("Cols in B: \n");
  scanf("%i", &cols_b);
  printf("Rows in B: \n");
  scanf("%i", &rows_b);

  //Validate if it's a posible operation
  if(rows_a != cols_b){
    printf("Number of rows in A must match cols in B\n");
    //Terminate program if operation is impossible
    return 0;
  }

  //Allocate memory in CPU
  matrix_a = (int *)malloc(sizeof(int) * cols_a * rows_a);
  matrix_b = (int *)malloc(sizeof(int) * cols_b * rows_b);
  result_matrix = (int *)malloc(sizeof(int) * cols_b * rows_a);

  //Allocate memory in device
  hipMalloc((void **)&d_matrix_a, sizeof(int) * cols_a * rows_a);
  hipMalloc((void **)&d_matrix_b, sizeof(int) * cols_b * rows_b);
  hipMalloc((void **)&d_result_matrix, sizeof(int) * cols_b * rows_a);

  //Get values of Matrix A
  printf("A = ");
  for (int i = 0; i < cols_a * rows_a; i ++ ){
    if (i%cols_a == 0){
      printf("\n");
    }
    printf("Enter value %i for matrix A: \n", i+1);
    scanf("%i", &aux_a);
    matrix_a[i] = aux_a;
  }

  //Get values of Matrix B
  printf("\nB = ");
  for (int i = 0; i < cols_b * rows_b; i ++ ){
    if (i%cols_a == 0){
      printf("\n");
    }
    printf("Enter value %i for matrix B: \n", i+1);
    scanf("%i", &aux_b);
    matrix_b[i] = aux_b;
  }

  //Copy values to device variables
  hipMemcpy(d_matrix_a, matrix_a, sizeof(int) * cols_a * rows_a, hipMemcpyHostToDevice);
  hipMemcpy(d_matrix_b, matrix_b, sizeof(int) * cols_b * rows_b, hipMemcpyHostToDevice);

  dim3 dimThreads(N_THREADS, 1, 1);
  dim3 dimBlocks(N_BLOCKS, 1, 1);

  //Execute global function in device
  multiplication<<<dimBlocks, dimThreads>>>(d_matrix_a, d_matrix_b, d_result_matrix, cols_a, rows_a, cols_b, rows_b);

  //Retrieve Result Matrix from device
  hipMemcpy(result_matrix, d_result_matrix, sizeof(int) * rows_a * cols_b, hipMemcpyDeviceToHost);

  //Print Result Matrix
  printf("\nR = ");
  for (int i = 0; i < cols_b * rows_a; i++){
    if (i%cols_b == 0){
      printf("\n");
    }
    printf("%i\t", result_matrix[i]);
  }
  printf("\n");

  //Free Memory
  free(matrix_a);
  free(matrix_b);
  free(result_matrix);
  hipFree(d_matrix_a);
  hipFree(d_matrix_b);
  hipFree(d_result_matrix);
}
