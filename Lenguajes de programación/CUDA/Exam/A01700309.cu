//Rodolfo Martinez Guevara
// template provided for cuda quizz 3.
// remember to write your own comments in the code below.


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 9		//size of original matrix
#define K N/3		//size of compressed matrrix
#define ThreadsPerBlock 1 		 // choose wisely
#define NumBlocks N  		// choose wisely

__global__ void compress(float *mat, int n, float *comp, int k){
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  int tidy = threadIdx.y + blockIdx.y * blockDim.y;
  int index = tidx + tidy * n;
  int aux = 0;

  int current_col = 0;
  int current_row = 0;

  if(index < n){
    current_col = index/k;
    current_row = index%n;

    for (int i = 0; i < k; i++){
      for (int j = 0; j < k; j++){
        aux += mat[i*(current_row*current_col)+j];
      }
    }
    comp[index] = aux;
  }
}

void print_mat(float *mat, int n){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			printf("%.1f\t", mat[i*n+j]);
		}
		printf("\n");
	}
	printf("\n");
}


void fill_mat(float *mat, int n){
	int c = 0;
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			mat[i*n+j] = c++;
		}
	}
}

int main(){
	float *h_compress, *h_matrix;
	float *d_compress, *d_matrix;

	h_compress = (float *)malloc(sizeof(float)*K*K);
	h_matrix = (float *)malloc(sizeof(float)*N*N);

  hipMalloc((void **)&d_matrix, sizeof(float)*N*N);
  hipMalloc((void **)&d_compress, sizeof(float)*K*K);

	fill_mat(h_matrix, N);
	// fill_mat(h_compress, K);

	printf("\n input mat \n");
	print_mat(h_matrix, N);

  hipMemcpy(d_matrix, h_matrix, sizeof(float)*N*N, hipMemcpyHostToDevice);

  dim3 dimThreads(ThreadsPerBlock, 1,1);
  dim3 dimBlocks(NumBlocks, 1,1);

  compress<<<dimBlocks, dimThreads>>>(d_matrix, N, d_compress, K);

  hipMemcpy(h_compress, d_compress, sizeof(float)*K*K, hipMemcpyDeviceToHost);

  printf("\n input compress \n");
  print_mat(h_compress, K);

  free(h_matrix);
  free(h_compress);
  hipFree(d_matrix);
  hipFree(d_compress);


}
